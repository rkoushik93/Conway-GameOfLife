
#include <hip/hip_runtime.h>
#include<stdio.h>

__global__ void computeFutureGen(int* current,int* future,int n){
        int col=threadIdx.x+blockIdx.x*blockDim.x;
        int row=threadIdx.y+blockIdx.y*blockDim.y;
        int index=col+row*n;
        //Computing the number of alive neighbors
        int neighAlive=0;
if(col<n && row<n){     //Computation starts only when the thread is within our matrix row & column limit
        //Different cases have to considered before counting the alive neighbors
        if(col==0 && row==0){   //When current node is at top left corner of the matrix
                neighAlive+=current[index+1];   //Neighbor to the right
                neighAlive+=current[index+n];   //Neighbor at the bottom
                neighAlive+=current[index+n+1]; //Neighbor at the bottom right diagonal
        }
        else if(col==0 && row==n-1){    //When current node is at bottom left corner
                neighAlive+=current[index+1];   //Neighbor to the right
                neighAlive+=current[index-n];   //Neighbor directly above
                neighAlive+=current[index-n+1]; //Neighbor at top right diagonal
        }
        else if(col==n-1 && row==0){    //When current node is at top right corner
                neighAlive+=current[index-1];   //Neighbor to the left
                neighAlive+=current[index+n-1]; //Neighbor at bottom left
                neighAlive+=current[index+n];   //Neighbor exactly below
        }
        else if(col==n-1 && row==n-1){  //When current node is at bottom right corner
                neighAlive+=current[index-1];   //Neighbor to the left
                neighAlive+=current[index-n];   //Neighbor exactly above
                neighAlive+=current[index-n-1]; //Neighbor at top left diagonally
        }
        else if(row==0 && col>0 && col<n-1){    //When current node is at top wall excluding the corners
                neighAlive+=current[index-1];   //Neighbor to the left
                neighAlive+=current[index+1];   //Neighbor to the right
                neighAlive+=current[index+n-1]; //Neighbor diagonally left below
                neighAlive+=current[index+n];   //Neighbor exactly below
                neighAlive+=current[index+n+1]; //Neighbor diagonally right below
        }
        else if(row==n-1 && col>0 && col<n-1){  //When current node is on bottom wall excluding the corners
                neighAlive+=current[index-1];   //Neighbor to the left
                neighAlive+=current[index+1];   //Neighbor to the right
                neighAlive+=current[index-n+1]; //Neighbor diagonally right on top
                neighAlive+=current[index-n];   //Neighbor exactly above
                neighAlive+=current[index-n-1]; //Neighbor diagonally left on top
        }
        else if(col==0 && row>0 && row<n-1){    //When current node is on left wall excluding corners
                neighAlive+=current[index+1];   //Neighbor to the right
                neighAlive+=current[index-n];   //Neighbor exactly above
                neighAlive+=current[index-n+1]; //Neighbor diagonally right on top
                neighAlive+=current[index+n];   //Neighbor exactly down
                neighAlive+=current[index+n+1]; //Neighbor diagonally right below
        }
        else if(col==n-1 && row>0 && row<n-1){  //When current node is on right wall excluding corners
                neighAlive+=current[index-1];   //Neighbor to the left
                neighAlive+=current[index-n];   //Neighbor exactly above
                neighAlive+=current[index-n-1]; //Neighbor diagonally left on top
                neighAlive+=current[index+n-1]; //Neighbor diagonally left below
                neighAlive+=current[index+n];   //Neighbor exactly below
        }
        else{   //For all middle elements, within the boundaries described above
                neighAlive+=current[index-1];   //Neighbor to the left
                neighAlive+=current[index+1];   //Neighbor to the right
                neighAlive+=current[index-n-1]; //Neighbor diagonally left on top
                neighAlive+=current[index-n];   //Neighbor exactly above
                neighAlive+=current[index-n+1]; //Neighbor diagonally right on top
                neighAlive+=current[index+n-1]; //Neighbor diagonally left below
                neighAlive+=current[index+n];   //Neighbor exactly below
                neighAlive+=current[index+n+1]; //Neighbor diagonally right below
        }



        //Code block to decide the alive status of a cell based on the number of alive neighbors
        if(current[index]==1 && neighAlive<2)
                future[index]=0;
        else if(current[index]==1 && (neighAlive==2 || neighAlive==3))
                future[index]=1;
        else if(current[index]==1 && neighAlive>3)
                future[index]=0;
        else if(current[index]==0 && neighAlive==3)
                future[index]=1;
        else
                future[index]=0;
}
}


int main(int argc,char** argv){
        int i,j,k;
        int n=0;
        n=atoi(argv[1]);
        int currentGen[n][n];
        int futureGen[n][n];
        dim3 threadsPerBlock(10,10);
        dim3 numBlocks(n/threadsPerBlock.x,n/threadsPerBlock.x);
        int* current;
        int* future;
        float milliseconds=0;
        hipError_t err;
        hipEvent_t start,stop;
        hipEventCreate(&start);
        hipEventCreate(&stop);

        /*Populating the input matrix currentGen with random 0's
        and 1's using rand() method*/
        for(i=0;i<n;i++)
                for(j=0;j<n;j++)
                        currentGen[i][j]=rand()%2;

        //Initializing the futureGen matrix with all 0's
        for(i=0;i<n;i++)
                for(j=0;j<n;j++)
                        futureGen[i][j]=0;

        //Allocating memory for device copy of current generation matrix
        hipMalloc((void **) &current,sizeof(int)*n*n);
        err=hipGetLastError();
        if(err!=hipSuccess)
                printf("\nERROR after hipMalloc of current : %s\n\n",hipGetErrorString(err));

        //Allocating memory for device copy of future generation matrix
        hipMalloc((void **) &future,sizeof(int)*n*n);
         err=hipGetLastError();
        if(err!=hipSuccess)
                printf("\nERROR after hipMalloc of future: %s\n\n",hipGetErrorString(err));


        //Copying current generation matrix from host to device
        hipMemcpy(current,currentGen,sizeof(int)*n*n,hipMemcpyHostToDevice);
        err=hipGetLastError();
        if(err!=hipSuccess)
                printf("\nERROR after hipMemcpy of currentGen to current: %s\n\n",hipGetErrorString(err));


        //Displaying the first 10 rows and columns of currentGen matrix
        printf("\nPrinting the alive state of first 10 rows and columns of %dx%d current generation matrix\n",n,n);
        for(i=0;i<10;i++){
                for(j=0;j<10;j++){
                        printf("%d\t",currentGen[i][j]);
                }
                printf("\n");
        }

        hipEventRecord(start);
        /*Loop for calculating the alive state of the cells after
        10, 100 and 1000 iterations*/
        for(k=1;k<=1000;k++){

        //Calling the kernel
        if(k==1)
                computeFutureGen<<<numBlocks,threadsPerBlock>>>(current,future,n);
        else
                computeFutureGen<<<numBlocks,threadsPerBlock>>>(future,future,n);

        err=hipGetLastError();
        if(err!=hipSuccess)
                printf("\nERROR after kernel call: %s\n\n",hipGetErrorString(err));
        hipEventRecord(stop);

        //Copying the result from device to host
        hipMemcpy(futureGen,future,sizeof(int)*n*n,hipMemcpyDeviceToHost);
        err=hipGetLastError();
        if(err!=hipSuccess)
                printf("\nERROR after hipMemcpy of future to futureGen: %s\n\n",hipGetErrorString(err));


        //Displaying the first 10 rows and columns of futureGen matrix
        //Display only after 10th, 100th and 1000th iteration
        if(k==10 || k==100 || k==1000){
        printf("\nPrinting the alive state of first 10 rows and columns of %dx%d future generation matrix after %d iterations\n",n,n,k);
        for(i=0;i<10;i++){
                for(j=0;j<10;j++){
                        printf("%d\t",futureGen[i][j]);
                }
                printf("\n");
        }
        hipEventElapsedTime(&milliseconds,start,stop);
        printf("Time taken for this computation = %f milliseconds\n\n",milliseconds);
        }

        }
        return 0;
}


